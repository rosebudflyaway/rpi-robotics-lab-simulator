#include "hip/hip_runtime.h"

#define MSH_WIDTH   150
#define MSH_HEIGHT  150

#define BLOCK_N 10

// Includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// OpenGL Graphics includes
#include <GL/glew.h>
#if defined (__APPLE__) || defined(MACOSX)
#include <GLUT/glut.h>
#else
#include <GL/freeglut.h>
#endif

// Includes, cudah
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <timer.h>               // timing functions

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <helper_cuda_gl.h>      // helper functions for CUDA/GL interop
#include "cuda_utils.cuh"

#include <hip/hip_vector_types.h>

#define MAX_EPSILON_ERROR 10.0f
#define THRESHOLD         0.30f
#define REFRESH_DELAY     0.0  //10    //ms
#define TIME_STEP         0.001 // s
#define MAX_FORCE         5.0   // N

////////////////////////////////////////////////////////////////////////////////
// Constants
const unsigned int window_width  = 512;
const unsigned int window_height = 512;

const unsigned int mesh_width    = MSH_WIDTH;//256;
const unsigned int mesh_height   = MSH_HEIGHT;//256;

int simulation_step = 0; 
time_t SimTimerStart;
time_t SimTimerStop;  

// VBO variables
GLuint vbo;
struct hipGraphicsResource *cuda_vbo_resource;
void *d_vbo_buffer = NULL;

float g_fAnim = 0.0;

// Mouse controls
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0, rotate_y = 0.0;
float translate_z = -3.0;

StopWatchInterface *timer = NULL;
StopWatchInterface *simTimer = NULL; 

// Auto-Verification Code
int fpsCount = 0;        // FPS count for averaging
int fpsLimit = 1;        // FPS limit for sampling
int g_Index = 0;
float avgFPS = 0.0f;
unsigned int frameCount = 0;
unsigned int g_TotalErrors = 0;
bool g_bQAReadback = false;

int *pArgc = NULL;
char **pArgv = NULL;

#define MAX(a,b) ((a > b) ? a : b)
#define MIN(a,b) ((a < b) ? a : b) 

////////////////////////////////////////////////////////////////////////////////
// Declaration, forward
bool runTest(int argc, char **argv, char *ref_file);
void cleanup();

// GL functionality
bool initGL(int *argc, char **argv);
void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
               unsigned int vbo_res_flags);
void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res);
bool animate = false; 

// Rendering callbacks
void display();
void keyboard(unsigned char key, int x, int y);
void mouse(int button, int state, int x, int y);
void motion(int x, int y);
void timerEvent(int value);

// CUDA functionality
void initCuda(struct hipGraphicsResource **vbo_resource); 
void runCuda(struct hipGraphicsResource **vbo_resource);
const int NUM_BLOCKS = 1024;
const int NUM_YBLOCKS = 10; 
const int THREADS_PER_BLOCK = MIN(1024, mesh_width*mesh_height); 
const char *sSDKsample = "CUDA Cloth Example";

// On device global memory for storing body (point) attributes
__device__ float3 point_velocities[mesh_width*mesh_height];
__device__ float3 point_forces[mesh_width*mesh_height]; 
__device__ bool point_is_fixed[mesh_width*mesh_height]; 
float3 *dev_pos; 
float3 *dev_forces;



///////////////////////////////////////////////////////////////////////////////
// Initialize the positions of all the cloth particles.  
///////////////////////////////////////////////////////////////////////////////
__global__ void initialize_particles(float4 *pos, unsigned int width, unsigned int height) {
    //unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;   
    //unsigned int y = tid / width; 
    //unsigned int x = tid % width; 
    
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int tid = y*MSH_WIDTH + x;
    
    //printf("INIT hello from %d\n", tid);
        //return;

    if (tid < width*height) {
        // Calculate uv coordinates (w = 0.0) 
        float u = x / (float) width;
        float v = y / (float) height;
        u = u*((float)width)/((float)width-1);
        v = v*((float)height)/((float)height-1);

        // Set vertex position
        pos[tid] = make_float4(u, 0.0f, v, 1.0f);
        
        // Initialize velocity
        point_velocities[tid] = make_float3(0.0, 0.0, 0.0); 
        
        // Fix the top corners
        //if ((x==0 || x==width-1) && y==height-1) 
        //if (x==0 && y==0 || x==0 && y==mesh_height-1 || x==mesh_width-1 && y==0 || x==mesh_width-1 && y==mesh_height-1 || x==mesh_width/2 && y==mesh_height/2)
        
        bool makeStatic = false;
        // corners
        if (x<4 && y==0 || x==0 && y<4 ||
            x > mesh_width-5 && y == 0 || x == mesh_width-1 && y < 4 ||
            x > mesh_width-5 && y == mesh_height-1 || x == mesh_width-1 && y > mesh_height-5 ||
            x < 4 && y == mesh_height-1 || x == 0 && y > mesh_height - 5)
            makeStatic = true;
        
        float unit = 410.0/mesh_width;
        float dx = abs(1.0*x-mesh_width/2.0);
        float dy = abs(1.0*y-mesh_height/2.0); 
        //if (makeStatic || dx < unit && dy < unit )
        if (makeStatic || x==y )
            point_is_fixed[tid] = true; 
        else
            point_is_fixed[tid] = false; 
    }
}


// Given indices p1 and p2, calculates the 
__device__ void update_forces(float4 *pos, int p1, int p2, float l0, float k) {

    //assert(p1 > 0 && p1 < 5*5);
    //assert(p2 > 0 && p2 < 5*5);   NOTE! This broke the crap out of this
    float4 P1 = pos[p1]; 
    float4 P2 = pos[p2];
    float3 L = make_float3( (P2.x-P1.x),        // Vector from p1 to p2
                            (P2.y-P1.y),
                            (P2.z-P1.z)  ); 
    //float l0 = 0.1;  // TODO: this should not be hard-coded 
    float dist = sqrt( L.x*L.x +            // Distance between points
                       L.y*L.y +
                       L.z*L.z   );
    float3 Fint = make_float3( k * (L.x - (l0/dist)*L.x),
                               k * (L.y - (l0/dist)*L.y),
                               k * (L.z - (l0/dist)*L.z)  );  
    
    atomicAddf(&point_forces[p1].x, Fint.x); 
    atomicAddf(&point_forces[p1].y, Fint.y);
    atomicAddf(&point_forces[p1].z, Fint.z);
    
    atomicAddf(&point_forces[p2].x, -Fint.x); 
    atomicAddf(&point_forces[p2].y, -Fint.y);
    atomicAddf(&point_forces[p2].z, -Fint.z);
    
    //printf("Updating %d and %d.  Dist: %f\n",p1,p2,dist);
}


// Here, we manually move particles back from constraint violation
// Assumes that p1 is not static. 
__device__ float4 getCorrection(float4 *pos, unsigned int p1, unsigned int p2, float l0) {
    float4 correction = make_float4(0.0, 0.0, 0.0, 0.0); 
    if (point_is_fixed[p1] && point_is_fixed[p2])
        return correction;
    float4 P1 = pos[p1]; 
    float4 P2 = pos[p2];
    float4 n = make_float4( (P2.x-P1.x),        // Vector from p1 to p2
                            (P2.y-P1.y),
                            (P2.z-P1.z), 0.0 ); 
  
    float l1 = length(n);                // Distance between points
    normalize(n); 
    
    float deltaP = l1 - (1.1) * l0;
    if (deltaP > 0) {
        if (point_is_fixed[p2])
            correction += deltaP * n;
        else
            correction += (deltaP/2.0) * n;
    }
    return correction; 
}

///////////////////////////////////////////////////////////////////////////////
// Simple kernel to modify vertex positions of the particles
///////////////////////////////////////////////////////////////////////////////
#define K_STRUCTURAL 5.0
#define K_SHEAR 2.0
#define K_BEND 2.0
#define DAMPING 0.05
__global__ void update_particles(float4 *pos, unsigned int width, unsigned int height, float time, float3 *dev_pos, float3 *dev_forces)
{
    //unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;     // Thread ID    
    //unsigned int y = tid / width;                               // x and y grid coordinates
    //unsigned int x = tid % width; 
    
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int tid = y*MSH_WIDTH + x;
    
    //printf("UPDATE hello from %d\n", tid);
        //return;

    if (tid < width*height) {       // Asserts that the thread points to a valid index
        // Initialize force as zero + gravity
        point_forces[tid] = make_float3(0.0, 0.0, 0.0); 
       
        __syncthreads();  // Don't allow changes until all forces have been reset
                
        // Update force for each neighbor particle
        // Structural springs
        float l0 = 1.0/(width-1);  
        
        if (x < width-1)                    // Structural springs
            update_forces( pos, tid, tid+1, l0, K_STRUCTURAL );         // 1-left
        if (y < height-1)
            update_forces( pos, tid, tid+width, l0, K_STRUCTURAL );     // 1-above
        if (x < width-1 && y < height-1)    // Shear springs X
            update_forces( pos, tid, tid+width+1, 1.41421356237*l0, K_SHEAR);  // up-left
        if (x < width-1 && y > 0)
            update_forces( pos, tid, tid-width+1, 1.41421356237*l0, K_SHEAR);  // down-left
        if (x < width-2)                    // Flexion springs + (2 away)
            update_forces( pos, tid, tid+2, 2*l0, K_BEND);              // 2-left
        if (y < height-2) 
            update_forces( pos, tid, tid+2*width, 2*l0, K_BEND);        // 2-above
        
        __syncthreads(); // Don't allow integration until forces are completely updated
        
        if (!point_is_fixed[tid]) {
            // Limit the maximum force on each particle
            point_forces[tid].x = MIN(point_forces[tid].x, MAX_FORCE);
            point_forces[tid].x = MAX(point_forces[tid].x, -MAX_FORCE);  
            point_forces[tid].y = MIN(point_forces[tid].y, MAX_FORCE);  
            point_forces[tid].y = MAX(point_forces[tid].y, -MAX_FORCE); 
            point_forces[tid].z = MIN(point_forces[tid].z, MAX_FORCE); 
            point_forces[tid].z = MAX(point_forces[tid].z, -MAX_FORCE); 
        
            // Kinematic update (include damping) 
            // This doesn't need to be atomic since we sync
            float mass = 0.1 / mesh_width*mesh_height; 
            atomicAddf(&point_forces[tid].x, mass *  0.0 - DAMPING * point_velocities[tid].x);
            atomicAddf(&point_forces[tid].y, mass * -9.8 - DAMPING * point_velocities[tid].y);
            atomicAddf(&point_forces[tid].z, mass *  0.0 - DAMPING * point_velocities[tid].z);    
            
            point_velocities[tid].x += (point_forces[tid].x/mass) * TIME_STEP;  
            point_velocities[tid].y += (point_forces[tid].y/mass) * TIME_STEP;
            point_velocities[tid].z += (point_forces[tid].z/mass) * TIME_STEP;
                                          
            // Integrate velocity over time step
            pos[tid].x += point_velocities[tid].x * TIME_STEP;
            pos[tid].y += point_velocities[tid].y * TIME_STEP;
            pos[tid].z += point_velocities[tid].z * TIME_STEP;    
        }
        
        __syncthreads();  // Interesting question, what would happen if this was inside the if() above
            
        for (int constraint_iter=0; constraint_iter<MAX(width,6); constraint_iter++) {
            
            float4 correction = make_float4(0.0, 0.0, 0.0, 0.0); 
            // Determine correction from constraint violation
            if (!point_is_fixed[tid]) {
                // Structural springs +
                if (x < width-1)                    
                    correction += getCorrection( pos, tid, tid+1, l0 );              // 1-left
                if (x > 0)                    
                    correction += getCorrection( pos, tid, tid-1, l0 );              // 1-right
                if (y < height-1)
                    correction += getCorrection( pos, tid, tid+width, l0 );          // 1-above
                if (y > 0)
                    correction += getCorrection( pos, tid, tid-width, l0 );          // 1-below
                // Shear springs X    
                if (x < width-1 && y < height-1)    
                    correction += getCorrection( pos, tid, tid+width+1, 1.41421356237*l0 ); // up-left   
                if (x > 0 && y < height-1) 
                    correction += getCorrection( pos, tid, tid+width-1, 1.41421356237*l0 ); // up-right
                if (x < width-1 && y > 0)
                    correction += getCorrection( pos, tid, tid-width+1, 1.41421356237*l0 ); // down-left
                if (x > 0 && y > 0)
                    correction += getCorrection( pos, tid, tid-width-1, 1.41421356237*l0 ); // down-right
            }
            
            __syncthreads(); 
            
            // Update position from correction
            pos[tid] += 0.95*correction; 
            
            __syncthreads(); 
            
            //printf("[%d] : (%f, %f, %f, %f)\n",tid,correction.x, correction.y, correction.z, correction.w);
        
        } // End of constraint correction iterations
        
        
        // Debug info
        dev_pos[tid].x = pos[tid].x;  //  TODO: this is for debugging
        dev_pos[tid].y = pos[tid].y;
        dev_pos[tid].z = pos[tid].z; 
        
        dev_forces[tid].x = point_forces[tid].x;
        dev_forces[tid].y = point_forces[tid].y;
        dev_forces[tid].z = point_forces[tid].z;
                 
    }
}


// Execute the kernel
void launch_kernel(float4 *pos, unsigned int mesh_width, unsigned int mesh_height, float time)
{    
    dim3 block(BLOCK_N, BLOCK_N, 1);
    dim3 grid(max(1,mesh_width / block.x), max(1,mesh_height / block.y), 1);
    
    //printf("BLOCK: %d, %d, %d\nGRID: %d, %d, %d\n",block.x, block.y, block.z, grid.x, grid.y, grid.z);
    
    update_particles<<<grid,block>>>(pos, mesh_width, mesh_height, time, dev_pos, dev_forces);
    //update_particles<<<NUM_BLOCKS,THREADS_PER_BLOCK>>>(pos, mesh_width, mesh_height, time, dev_pos, dev_forces);
    
    //printf("Time step: %d\n",simulation_step);
    if (simulation_step == 0) {
        printf("Starting simulation stopwatch...\n");
        sdkCreateTimer(&simTimer); 
        sdkStartTimer(&simTimer);
    }
    else if (simulation_step == 1000) {
        sdkStopTimer(&simTimer); 
        float seconds = sdkGetTimerValue(&simTimer) / 1000.0;
        printf("Time taken for first 1000 simulation steps of %d particles: %f seconds.\n", mesh_width*mesh_height, seconds);
    }
    simulation_step++; 
}



////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    // TODO
    hipMalloc( (void**) &dev_pos, mesh_width*mesh_height*sizeof(float3) );
    hipMalloc( (void**) &dev_forces, mesh_width*mesh_height*sizeof(float3) ); 

    char *ref_file = NULL;

    pArgc = &argc;
    pArgv = argv;

    printf("%s starting...\n", sSDKsample);

    runTest(argc, argv, ref_file);

    hipDeviceReset();
    printf("%s completed, returned %s\n", sSDKsample, (g_TotalErrors == 0) ? "OK" : "ERROR!");
    exit(g_TotalErrors == 0 ? EXIT_SUCCESS : EXIT_FAILURE);
}

void computeFPS()
{
    frameCount++;
    fpsCount++;

    if (fpsCount == fpsLimit)
    {
        avgFPS = 1.f / (sdkGetAverageTimerValue(&timer) / 1000.f);
        fpsCount = 0;
        fpsLimit = (int)MAX(avgFPS, 1.f);

        sdkResetTimer(&timer);
    }

    char fps[256];
    sprintf(fps, "Cuda GL Interop (VBO): %3.1f fps (Max 100Hz)", avgFPS);
    glutSetWindowTitle(fps);
}

////////////////////////////////////////////////////////////////////////////////
//! Initialize GL
////////////////////////////////////////////////////////////////////////////////
bool initGL(int *argc, char **argv)
{
    glutInit(argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowSize(window_width, window_height);
    glutCreateWindow("Cuda GL Interop (VBO)");
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutMotionFunc(motion);
    glutTimerFunc(REFRESH_DELAY, timerEvent,0);

    // initialize necessary OpenGL extensions
    glewInit();

    if (! glewIsSupported("GL_VERSION_2_0 "))
    {
        fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
        fflush(stderr);
        return false;
    }

    // default initialization
    glClearColor(0.0, 0.0, 0.0, 1.0);
    glDisable(GL_DEPTH_TEST);

    // viewport
    glViewport(0, 0, window_width, window_height);

    // projection
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(60.0, (GLfloat)window_width / (GLfloat) window_height, 0.1, 100.0);

    SDK_CHECK_ERROR_GL();

    return true;
}


////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
bool runTest(int argc, char **argv, char *ref_file)
{
    // Create the CUTIL timer
    sdkCreateTimer(&timer);

    // First initialize OpenGL context, so we can properly set the GL for CUDA.
    // This is necessary in order to achieve optimal performance with OpenGL/CUDA interop.
    initGL(&argc, argv);

    // register callbacks
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutMouseFunc(mouse);
    glutMotionFunc(motion);

    // create VBO
    createVBO(&vbo, &cuda_vbo_resource, cudaGraphicsMapFlagsWriteDiscard);

    // run the cuda part
    printf("Run the cuda part\n"); 
    initCuda(&cuda_vbo_resource); 
    runCuda(&cuda_vbo_resource);

    // start rendering mainloop
    glutMainLoop();
    atexit(cleanup);
   

    return true;
}

////////////////////////////////////////////////////////////////////////////////
// INIT the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void initCuda(struct hipGraphicsResource **vbo_resource)
{
    // map OpenGL buffer object for writing from CUDA
    float4 *dptr;
    hipGraphicsMapResources(1, vbo_resource, 0);
    size_t num_bytes;
    hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes, *vbo_resource);
    printf("CUDA mapped VBO: May access %ld bytes\n", num_bytes);
    
    //dim3 block(NUM_BLOCKS, THREADS_PER_BLOCK, 1);
    //dim3 grid(max(1,mesh_width / block.x), max(1,mesh_height / block.y), 1);
    
    dim3 block(BLOCK_N, BLOCK_N, 1);
    dim3 grid(max(1,mesh_width / block.x), max(1,mesh_height / block.y), 1);
    initialize_particles<<<grid,block>>>(dptr, mesh_width, mesh_height);
    
    //initialize_particles<<<NUM_BLOCKS,THREADS_PER_BLOCK>>>(dptr, mesh_width, mesh_height);
        
    launch_kernel(dptr, mesh_width, mesh_height, g_fAnim);
    // unmap buffer object
    checkCudaErrors(hipGraphicsUnmapResources(1, vbo_resource, 0));
}


////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void runCuda(struct hipGraphicsResource **vbo_resource)
{
    if (animate) {
        // map OpenGL buffer object for writing from CUDA
        float4 *dptr;
        hipGraphicsMapResources(1, vbo_resource, 0);
        size_t num_bytes;
        hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes, *vbo_resource);
        //printf("CUDA mapped VBO: May access %ld bytes\n", num_bytes);
            
        for (int i=0; i<10; i++) // Simulate 10 steps before updating graphics
            launch_kernel(dptr, mesh_width, mesh_height, g_fAnim);

        // unmap buffer object
        checkCudaErrors(hipGraphicsUnmapResources(1, vbo_resource, 0));
        // The reason we map and unmap every time is for synchronicity 
    }
}


////////////////////////////////////////////////////////////////////////////////
//! Create VBO
////////////////////////////////////////////////////////////////////////////////
void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
               unsigned int vbo_res_flags)
{
    assert(vbo);

    // create buffer object
    glGenBuffers(1, vbo);
    glBindBuffer(GL_ARRAY_BUFFER, *vbo);

    // initialize buffer object
    unsigned int size = mesh_width * mesh_height * 4 * sizeof(float);
    glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);

    glBindBuffer(GL_ARRAY_BUFFER, 0);

    // register this buffer object with CUDA
    hipGraphicsGLRegisterBuffer(vbo_res, *vbo, vbo_res_flags);

    SDK_CHECK_ERROR_GL();
}

////////////////////////////////////////////////////////////////////////////////
//! Delete VBO
////////////////////////////////////////////////////////////////////////////////
void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res)
{

    // unregister this buffer object with CUDA
    hipGraphicsUnregisterResource(vbo_res);

    glBindBuffer(1, *vbo);
    glDeleteBuffers(1, vbo);

    *vbo = 0;
}

////////////////////////////////////////////////////////////////////////////////
//! Display callback
////////////////////////////////////////////////////////////////////////////////
void display()
{
    sdkStartTimer(&timer);

    // run CUDA kernel to generate vertex positions
    runCuda(&cuda_vbo_resource);

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    // set view matrix
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glTranslatef(0.0, 0.0, translate_z);
    glRotatef(rotate_x, 1.0, 0.0, 0.0);
    glRotatef(rotate_y, 0.0, 1.0, 0.0);

    // render from the vbo
    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    glVertexPointer(4, GL_FLOAT, 0, 0);

    // Draw points
    glPointSize(3.0); 
    glEnableClientState(GL_VERTEX_ARRAY);
        glColor3f(0.3, 0.3, 0.4);
        glDrawArrays(GL_POINTS, 0, mesh_width * mesh_height);
        //glDrawArrays(GL_TRIANGLES, 0, 300); 
    glDisableClientState(GL_VERTEX_ARRAY);
    
    // Draw axes
    // X axis
    glLineWidth(1.0); 
    glColor3f(1.0, 0.0, 0.0);
    glBegin(GL_LINES);
        glVertex3f(0.0, 0.0, 0.0);
        glVertex3f(1.0, 0.0, 0.0);
    glEnd();
    // Z axis
    glColor3f(0.0, 1.0, 0.0);
    glBegin(GL_LINES);
        glVertex3f(0.0, 0.0, 0.0);
        glVertex3f(0.0, 0.0, 1.0);
    glEnd();
    
    /*
    // Draw forces (REALLY INEFFICIENTLY)
    float3 host_pos[mesh_width*mesh_height];
    float3 host_force[mesh_width*mesh_height]; 
    
    hipMemcpy( host_pos, dev_pos, mesh_width*mesh_height*sizeof(float3), hipMemcpyDeviceToHost); 
    hipMemcpy( host_force, dev_forces, mesh_width*mesh_height*sizeof(float3), hipMemcpyDeviceToHost); 
    hipMemcpy( host_force, point_forces, mesh_width*mesh_height*sizeof(float3), hipMemcpyDeviceToHost);  
    
    glColor3f(0.0, 1.0, 0.0);
    float forceScale = 0.2; 
    for (int i=0; i<mesh_width*mesh_height; i++) {
        float3 p = host_pos[i];
        float3 f = forceScale*host_force[i];
        glBegin(GL_LINES);
            glVertex3f(p.x, p.y, p.z);
            glVertex3f(p.x+f.x, p.y+f.y, p.z+f.z);
        glEnd();
    }
    */
    
    glutSwapBuffers();

    if (animate)
        g_fAnim += 0.01f;

    sdkStopTimer(&timer);
    computeFPS();
}

void timerEvent(int value)
{
    glutPostRedisplay();
    glutTimerFunc(REFRESH_DELAY, timerEvent,0);
}

void cleanup()
{
    sdkDeleteTimer(&timer);

    if (vbo)
    {
        deleteVBO(&vbo, cuda_vbo_resource);
    }
}


////////////////////////////////////////////////////////////////////////////////
//! Keyboard events handler
////////////////////////////////////////////////////////////////////////////////
void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
    switch (key)
    {
        case (27) :
            exit(EXIT_SUCCESS);
            break;
        case (113) :
            exit(EXIT_SUCCESS);
            break;
        case (32) : 
            animate = !animate; 
            break;
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Mouse event handlers
////////////////////////////////////////////////////////////////////////////////
void mouse(int button, int state, int x, int y)
{
    if (state == GLUT_DOWN)
    {
        mouse_buttons |= 1<<button;
    }
    else if (state == GLUT_UP)
    {
        mouse_buttons = 0;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}

void motion(int x, int y)
{
    float dx, dy;
    dx = (float)(x - mouse_old_x);
    dy = (float)(y - mouse_old_y);

    if (mouse_buttons & 1)
    {
        rotate_x += dy * 0.2f;
        rotate_y += dx * 0.2f;
    }
    else if (mouse_buttons & 4)
    {
        translate_z += dy * 0.01f;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}



