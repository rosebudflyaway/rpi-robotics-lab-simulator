
// Attempt to code projected Gauss-Seidel using CUDA

#include <hip/hip_runtime.h>
#include <stdio.h>
#include "hipblas.h"

#define imin(a,b) (a<b?a:b)

//const int N = 33 * 1024;
//const int N = 200; 
const int N = 10;
const int threadsPerBlock = 256;
const int blocksPerGrid = imin( 32, (N+threadsPerBlock-1) / threadsPerBlock );

/////////////////////////////////////////////////
// Kernel functions

// adds to vectors a & b of length N, storing result in c
__global__ void add( int *a, int *b, int *c, int N ) {
    int tid = blockIdx.x;
    if (tid < N)
        c[tid] = a[tid] + b[tid];
}

// dot product of two vectors a and b, each of length N
__global__ void dot1( float *a, float *b, float *c, int N ) {
    __shared__ float cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;
    float temp = 0;
    while (tid < N) {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x; 
    }
    
    cache[cacheIndex] = temp; 
    
    __syncthreads(); 
    
    int i = blockDim.x/2;
    while (i != 0 ) {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }
    
    if (cacheIndex == 0)
        c[blockIdx.x] = cache[0];
        
}

/*
// float version of atomic add
__device__  float atomicAddf(float* address, float val) {
   float old = *address, assumed;

   do { assumed = old;
        old = __int_as_float( 
            atomicCAS((unsigned int*)address,
            __float_as_int(assumed),
            __float_as_int(val + assumed)) );
   } while (assumed != old);

   return old;
} 


__global__ void dot2( float *a, float *b, float *c, int N ) {
    __shared__ float temp[threadsPerBlock];
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    temp[threadIdx.x] = a[index] * b[index];
    //c[0] = 0; 
    __syncthreads();
    if( 0 == threadIdx.x ) {
        float sum = 0;
        for( int i = 0; i < N; i++ )
            sum += temp[i];
            atomicAddf( c , sum );
    }
}
*/

// A specialized matrix_row * vector multiply 
// Returns the multiplication of row in A times x
__device__ float MatVecMultiply(float *A, float *x, int N, int row) {
    float sum = 0;
    for (int i=0; i<N; i++) 
        sum += A[N*row+i] * x[i];
    return sum; 
}

// Projected Gauss-Seidel
// For a matrix A, vectors b, and x0, all of length N, iterate  
__global__ void pGS( float *A, float *b, float *x0, float *x1, int N ) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    float LAMBDA = 0.707;   // LAMBDA is supposed to be 1<=L<=2 but I think that's for positive(b)? 
                            // I wonder if for this method, LAMBDA should be dynamic with the size N 
    float r0, r1;
    
    // First iteration
    r0 = b[tid] + MatVecMultiply(A, x0, N, tid);
    //x1[tid] = max(0, x0[tid] - ri / A[N*tid+tid]);  // Projection  
    x1[tid] = x0[tid] - LAMBDA*r0 / A[N*tid+tid];  // Successive Over-Relaxation (SOR), no projection
    __syncthreads();  
    
    for (int iters=0; iters<100; iters++) {
        r1 = b[tid] + MatVecMultiply(A, x1, N, tid); 
        x0[tid] = x1[tid] - LAMBDA*r1 / A[N*tid+tid];  // Successive Over-Relaxation (SOR), no projection
        __syncthreads();  
        
        if (tid == 0) {
            printf("(%d) r = %f   : ",iters, r1);
            for (int i=0; i<N; i++) 
                printf("%f, \t",x0[i]);
            printf("\n"); 
        }
        __syncthreads(); 
        
        r0 = b[tid] + MatVecMultiply(A, x0, N, tid); 
        x1[tid] = x0[tid] - LAMBDA*r0 / A[N*tid+tid];  // Successive Over-Relaxation (SOR), no projection
        __syncthreads();  
        
        if (tid == 0)
            printf("   (%d) residuals = %f, %f\n",iters,r1, r0);
            
        if (abs(r0) < 0.001)
            break; 
    }
    
    
}


// Main
int main() {

    float *x1, *a, c, *partial_c;
    float *dev_A, *dev_x0, *dev_x1, *dev_a, *dev_b, *dev_partial_c;
    
    //A = (float*)malloc( N*N*sizeof(float) );
    x1 = (float*)malloc( N*sizeof(float) );
    a = (float*)malloc( N*sizeof(float) );
    //b = (float*)malloc( N*sizeof(float) );
    
    partial_c = (float*)malloc( blocksPerGrid*sizeof(float) );
    
    hipMalloc( (void**) &dev_A, N*N*sizeof(float) );
    hipMalloc( (void**) &dev_x0, N*sizeof(float) );
    hipMalloc( (void**) &dev_x1, N*sizeof(float) );
    hipMalloc( (void**) &dev_a, N*sizeof(float) );
    hipMalloc( (void**) &dev_b, N*sizeof(float) );
    hipMalloc( (void**) &dev_partial_c, blocksPerGrid*sizeof(float) );
    
    // Test case
    float A[100] ={ 38.2883,   2.2700,    3.2359,    3.6405,    1.7018,    2.6147,    2.6089,    3.5436,    3.2964,    2.7359,
                    2.2700,   22.3140,    2.4651,    2.3220,    1.2521,    2.2821,    1.8419,    2.4564,    2.1646,    2.4552,
                    3.2359,    2.4651,   46.1522,    3.9220,    2.2500,    3.8552,    3.2061,    4.0181,    3.4000,    4.2448,
                    3.6405,    2.3220,    3.9220,   52.2668,    2.3026,    3.2100,    3.0781,    4.1019,    3.9669,    3.8830,
                    1.7018,    1.2521,    2.2500,    2.3026,   13.5953,    1.8844,    1.6628,    2.2282,    2.0793,    2.3255,
                    2.6147,    2.2821,    3.8552,    3.2100,    1.8844,   36.4673,    2.5989,    3.7060,    2.5412,    3.7806,
                    2.6089,    1.8419,    3.2061,    3.0781,    1.6628,    2.5989,   25.4442,    3.1507,    2.5339,    2.7645,
                    3.5436,    2.4564,    4.0181,    4.1019,    2.2282,    3.7060,    3.1507,   46.4464,    3.2219,    3.8510,
                    3.2964,    2.1646,    3.4000,    3.9669,    2.0793,    2.5412,    2.5339,    3.2219,   43.1067,    3.7971,
                    2.7359,    2.4552,    4.2448,    3.8830,    2.3255,    3.7806,    2.7645,    3.8510,    3.7971,   49.5411 };
    float b[10] = { 9.838233, 1.661702, 6.239759, 2.908003, 0.016799, 4.226084, 2.732327, 5.940454, 3.192733, 2.837930  }; 
    for (int i=0; i<N; i++) b[i] = -b[i]; 
    float x0[10] = { 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0 }; 
    
    hipMemcpy( dev_A, A, N*N*sizeof(float), hipMemcpyHostToDevice ); 
    hipMemcpy( dev_b, b, N*sizeof(float), hipMemcpyHostToDevice ); 
    hipMemcpy( dev_x0, x0, N*sizeof(float), hipMemcpyHostToDevice ); 
    
    pGS<<<1,N>>>( dev_A, dev_b, dev_x0, dev_x1, N ); 
    
    hipMemcpy( x1, dev_x1, N*sizeof(float), hipMemcpyDeviceToHost );
    
    printf("Solution x = ");
    for (int i=0;i<N;i++)
        printf("%f, ",x1[i]);
    printf("\n");
    
    /*
    for (int i=0; i<N; i++) {
        a[i] = i;
        b[i] = i*2; 
    }
    float result = 0;
    for (int i=0; i<N; i++) {
        result += a[i]*b[i];
    }
    printf("Executing %d length dot product on %d blocks per grid with %d threads per block\n",N,blocksPerGrid,threadsPerBlock);
    printf("CPU result: %f\n",result);
    cudaMemcpy( dev_a, a, N*sizeof(float), cudaMemcpyHostToDevice ); 
    cudaMemcpy( dev_b, b, N*sizeof(float), cudaMemcpyHostToDevice );
    dot2<<<blocksPerGrid,threadsPerBlock>>>( dev_a, dev_b, dev_partial_c, N );
    cudaMemcpy( partial_c, dev_partial_c, blocksPerGrid*sizeof(float), cudaMemcpyDeviceToHost ); 
    printf(" The first index: %f\n",partial_c[0]);
    c = 0;
    for (int i=0; i<blocksPerGrid; i++) {
        c += partial_c[i];
    } 
    printf( "The dot product is: %f\n", c );
    */ 
    
    hipFree( dev_A );
    hipFree( dev_x0 );
    hipFree( dev_x1 );
    hipFree( dev_a );
    hipFree( dev_b );
    hipFree( dev_partial_c );
    
    //free( A );
    //free( x0 );
    free( x1 ); 
    free( a );
    //free( b );
    free( partial_c ); 
    
    return 0; 

}

