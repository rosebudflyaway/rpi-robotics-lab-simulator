#include "hip/hip_runtime.h"
/* 
 * File:   CollisionDetectionGPU.cpp
 * Author: Ying Lu
 * GPU Extension by Dave Kotfis
 * 
 * Created on November 11, 2014
 */


#include "CollisionDetectionGPU.h"

#include <hip/hip_runtime.h>

#define GROUND -1
#define SPHERE  1

__host__ __device__ glm::vec3 arbitrary_tangent(glm::vec3 n) {

  glm::vec3 Y = glm::vec3(0.0);   Y[1] = 1.0;
  glm::vec3 Z = glm::vec3(0.0);   Z[2] = 1.0;
  glm::vec3 T;

  if (abs(glm::dot(n, Z)) < 0.7)
    T = glm::cross(n, Z);
  else
    T = glm::cross(n, Y);

  return glm::normalize(T);
}

__host__ __device__ void collide_sphere_plane(Sphere sphere, int side, Contact contact) {

}

__device__ void collide_sphere_sphere(Sphere sphere1, Sphere sphere2, Contact contact) {
  if ( sphere1.IsStaticBody && sphere2.IsStaticBody)
    return; 
  double Radius1 = sphere1.Radius;
  double Radius2 = sphere2.Radius;
  glm::vec3 U1 = sphere1.U;
  glm::vec3 U2 = sphere2.U;
            
  // Check bounding sphere
  glm::vec3 n = U2 - U1;
  double n_norm = glm::length(n);
  double psi = 0.0;
  psi = n_norm - Radius1 - Radius2; 
  if ( psi< sphere1.Bounding_Radius - Radius1 + sphere2.Bounding_Radius - Radius2 ) {
    n = glm::normalize(n);

    glm::vec3 t = arbitrary_tangent(n);
    glm::vec3 r1 =  n*(float)Radius1;
    glm::vec3 r2 = -n*(float)Radius2;

    // update the new contact structure here
    //contact.contact_ID = cID;
    contact.body1_type = SPHERE;
    contact.body2_type = SPHERE;
    //contact.body1 = sphere1;
    //contact.body2 = sphere2;
    contact.normal = n;
    contact.tangent = t;
    contact.r1 = r1;
    contact.r2 = r2;
    contact.psi = psi;
    //cID++;

    if ( !sphere1.IsStaticBody ) 
      {
        sphere1.ContactCount++;  
        if (sphere1.BodyIndex < 0) {
          //sphere1.BodyIndex = bID;
          //bID++;
        }
      }
    if ( !sphere2.IsStaticBody ) 
      {
        sphere2.ContactCount++;  
        if (sphere2.BodyIndex < 0) {
          //sphere2.BodyIndex = bID;
          //bID++;
        }
      }
  }
}

__global__ void collide_spheres(Sphere* device_spheres, Contact* device_contacts, int num_spheres) {
  //TODO; How can we improve memory coherency here?
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int s1 = idx / num_spheres;
  int s2 = idx % num_spheres;

  if (s1 < num_spheres && s2 < num_spheres) {
    collide_sphere_sphere(device_spheres[s1], device_spheres[s2], device_contacts[idx]);
  }
}

extern "C" void findContactsGPU(Contact* contacts, Sphere* spheres, int num_contacts, int num_spheres) 
{
    int cID = 0;         // The contact ID 
    int bID = 1;         // The body ID, for indexing; NOTE: here bID starts with 1

    //Package up data and send to the GPU
    Contact* device_contacts;
    hipMalloc((void**)&device_contacts, num_contacts*sizeof(Contact));

    Sphere* device_spheres;
    hipMalloc((void**)&device_spheres, num_spheres*sizeof(Sphere));
    hipMemcpy(device_spheres, spheres, num_spheres*sizeof(Sphere), hipMemcpyHostToDevice);

    // Init body ids
    for (int s=0; s<num_spheres; s++ ) 
    {  
        spheres[s].BodyIndex = -1; 
        spheres[s].ContactCount = 0;
    }
    
    // SPHERE-GROUND collision detection
    for (int s = 0; s < num_spheres; s++) 
    {
        if (!spheres[s].IsStaticBody) 
        {

            glm::vec3 U = spheres[s].U;
            double Radius = spheres[s].Radius;
            double psi = U[2] - Radius; // psi=height-radius
            if (psi > 0.5)
                continue; 
            glm::vec3 n = glm::vec3(0.0);
            n[2] = 1.0;               // Normal to sphere is always up (+z direction)
            glm::vec3 t = arbitrary_tangent(n);

            glm::vec3 r1 = n;                                                           // r1 for ground
            glm::vec3 r2 = (float)(-1.0*Radius) * n;
            Contact c; 

            // update the new contact structure here
            c.contact_ID = cID;
            c.body1_type = GROUND;
            c.body2_type = SPHERE;
            c.body1 = -250;  // for ground 
            c.body2 = s;
            c.normal = n;
            c.tangent = t;
            c.r1 = r1;
            c.r2 = r2;
            c.psi = psi;
            cID++;

            //contacts.push_back(c);
            num_contacts++;
            spheres[s].ContactCount++; 

            if (spheres[s].BodyIndex < 0) 
            {
                spheres[s].BodyIndex = bID;
                bID++;
            }
        } // if dynamic
    } // for loop
    
    // SPHERE-LEFT-RIGHT (in box) collision detection
    for (int s = 0; s < num_spheres; s++) 
    {
        if (!spheres[s].IsStaticBody) 
        {
            glm::vec3 U = spheres[s].U;
            double Radius = spheres[s].Radius;
            double left_gap = 0.0;   double right_gap = 0.0;  double psi = 0.0;
            glm::vec3 n = glm::vec3(0.0);
            left_gap = U[1] - Radius + 2.0;    // left_bound - (-2) 
            right_gap = 2.0 - (U[1] + Radius); // 2 - right_bound

            if(left_gap > 0.5 && right_gap > 0.5)
                continue;
            if(left_gap < 0.5) {
                psi = left_gap;
                n[1] = 1.0;
            }
            else {
                psi = right_gap;
                n[1] = -1.0;
            }
            glm::vec3 t = arbitrary_tangent(n);
            glm::vec3 r1 = n; 
            glm::vec3 r2 = (float)(-1.0*Radius) * n;

            Contact c;
            // update the new contact structure here
            c.contact_ID = cID;
            c.body1_type = GROUND;
            c.body2_type = SPHERE;
            c.body1 = -250;  // for ground 
            c.body2 = s;
            c.normal = n;
            c.tangent = t;
            c.r1 = r1;
            c.r2 = r2;
            c.psi = psi;
            cID++;

            //contacts[num_contacts++] = c;
            //contacts.push_back(c);
            num_contacts++;
            spheres[s].ContactCount++; 
            
            if (spheres[s].BodyIndex < 0) 
            {
                spheres[s].BodyIndex = bID;
                bID++;
            }

        } // if 
    } // for loop
    
    // SPHERE-FRONT-BACK (in box) collision detection
    for (int s = 0; s < num_spheres; s++) 
    {
        if (!spheres[s].IsStaticBody) 
        {
            glm::vec3 U = spheres[s].U;
            double Radius = spheres[s].Radius;
            double back_gap = 0.0;   double front_gap = 0.0;  double psi = 0.0;
            glm::vec3 n = glm::vec3(0.0);
            back_gap = U[0] - Radius + 2.0;     // back_bound - (-2) 
            front_gap = 2.0 - (U[0] + Radius); // 2 - front_bound

            if(back_gap > 0.5 && front_gap > 0.5)
                continue;
            if(back_gap < 0.5) {
                psi = back_gap;
                n[0] = 1.0;
            }
            else {
                psi = front_gap;
                n[0] = -1.0;
            }

            glm::vec3 t = arbitrary_tangent(n);
            glm::vec3 r1 = n; 
            glm::vec3 r2 = (float)(-1*Radius) * n;

            Contact c;
            // update the new contact structure here
            c.contact_ID = cID;
            c.body1_type = GROUND;
            c.body2_type = SPHERE;
            c.body1 = -250;  // for ground 
            c.body2 = s;
            c.normal = n;
            c.tangent = t;
            c.r1 = r1;
            c.r2 = r2;
            c.psi = psi;
            cID++;

            //contacts.push_back(c);
            num_contacts++;
            spheres[s].ContactCount++; 

            if (spheres[s].BodyIndex < 0) 
            {
                spheres[s].BodyIndex = bID;
                bID++;
            }
        }  // if 
    } // for

    // SPHERE-SPHERE collision detection
    // TODO: Replace loops with kernel call to collide_spheres
    for (int s1=0; s1<num_spheres; s1++) 
    {
        for (int s2=s1+1; s2<num_spheres; s2++) 
        {
          Contact c;
          //collide_sphere_sphere(spheres[s1], spheres[s2], c);

          //contacts.push_back(c);
          num_contacts++;

        } // inward for loop
    } // outward for loop

    //Copy contact data back from GPU
    hipMemcpy(contacts, device_contacts, num_contacts*sizeof(Contact), hipMemcpyDeviceToHost);

    //Cleanup CUDA
    hipFree(device_spheres);
    hipFree(device_contacts);

} // end of function findContacts


